/*
*   This program is free software: you can redistribute it and/or modify
*   it under the terms of the GNU General Public License as published by
*   the Free Software Foundation, version 2.1
*
*   This program is distributed in the hope that it will be useful,
*   but WITHOUT ANY WARRANTY; without even the implied warranty of
*   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
*   GNU General Public License, version 2.1, for more details.
*
*   You should have received a copy of the GNU General Public License
*
*/

#include <jni.h>

#include "bb/segments/kv/bb_segsort.hpp"
#include "bb/segments/k/bb_segsort.hpp"
#include "bb/matrix/kv/bb_segsort.hpp"
#include "bb/matrix/k/bb_segsort.hpp"

#include "datatype.h"


template <class K>
int sort_matrix(K* key, int rows, int cols, const bb::k::SortContext<K>* context) {
  hipError_t err;
  K    *key_d;

  err = hipMalloc((void**)&key_d, sizeof(K) * rows * cols);
  CUDA_CHECK(err, "matrix: alloc key_d");

  err = hipMemcpy(key_d, key, sizeof(K) * rows * cols, hipMemcpyHostToDevice);
  CUDA_CHECK(err, "matrix: copy to key_d");

  bb::matrix::k::bb_segsort(key_d, rows, cols, context);

  err = hipMemcpy(key, key_d, sizeof(K) * rows * cols, hipMemcpyDeviceToHost);
  CUDA_CHECK(err, "matrix: copy to key_d");

  err = hipFree(key_d);
  CUDA_CHECK(err, "matrix: free key_d");

  return 0;
}

template <class K, class V>
int sort_matrix(K* key, V* val, int rows, int cols, const bb::kv::SortContext<K, V>* context) {
  hipError_t err;
  K    *key_d;
  V    *val_d;

  err = hipMalloc((void**)&key_d, sizeof(K) * rows * cols);
  CUDA_CHECK(err, "matrix: alloc key_d");
  err = hipMalloc((void**)&val_d, sizeof(V) * rows * cols);
  CUDA_CHECK(err, "matrix: alloc val_d");

  err = hipMemcpy(key_d, key, sizeof(K) * rows * cols, hipMemcpyHostToDevice);
  CUDA_CHECK(err, "matrix: copy to key_d");
  err = hipMemcpy(val_d, val, sizeof(V) * rows * cols, hipMemcpyHostToDevice);
  CUDA_CHECK(err, "matrix: copy to val_d");

  bb::matrix::kv::bb_segsort(key_d, val_d, rows, cols, context);

  err = hipMemcpy(key, key_d, sizeof(K) * rows * cols, hipMemcpyDeviceToHost);
  CUDA_CHECK(err, "matrix: copy to key_d");
  err = hipMemcpy(val, val_d, sizeof(K) * rows * cols, hipMemcpyDeviceToHost);
  CUDA_CHECK(err, "matrix: copy to val_d");

  err = hipFree(key_d);
  CUDA_CHECK(err, "matrix: free key_d");
  err = hipFree(val_d);
  CUDA_CHECK(err, "matrix: free val_d");

  return 0;
}

template <class K>
int sort_matrix(K* key, int rows, int cols) {
  bb::k::SortContext<K> context_k(rows * cols, rows);
  return sort_matrix(key, rows, cols, &context_k);
}

template <class K, class V>
int sort_matrix(K* key, V* val, int rows, int cols) {
  bb::kv::SortContext<K, V> context_kv(rows * cols, rows);
  return sort_matrix(key, val, rows, cols, &context_kv);
}


template <class K>
int sort_segments(K* key, int* seg, int n, int length, const bb::k::SortContext<K>* context) {
  hipError_t err;
  K    *key_d;
  int  *seg_d;

  err = hipMalloc((void**)&key_d, sizeof(K)*n);
  CUDA_CHECK(err, "segments: alloc key_d");
  err = hipMalloc((void**)&seg_d, sizeof(int)*length);
  CUDA_CHECK(err, "segments: alloc seg_d");

  err = hipMemcpy(key_d, key, sizeof(K)*n, hipMemcpyHostToDevice);
  CUDA_CHECK(err, "segments: copy to key_d");
  err = hipMemcpy(seg_d, seg, sizeof(int)*length, hipMemcpyHostToDevice);
  CUDA_CHECK(err, "segments: copy to seg_d");

  bb::segments::k::bb_segsort(key_d, n, seg_d, length, context);

  err = hipMemcpy(key, key_d, sizeof(K)*n, hipMemcpyDeviceToHost);
  CUDA_CHECK(err, "segments: copy to key_d");

  err = hipFree(key_d);
  CUDA_CHECK(err, "segments: free key_d");
  err = hipFree(seg_d);
  CUDA_CHECK(err, "segments: free seg_d");

  return 0;
}

template <class K, class V>
int sort_segments(K* key, V* val, int* seg, int n, int length, const bb::kv::SortContext<K, V>* context) {
  hipError_t err;
  K    *key_d;
  V    *val_d;
  int  *seg_d;

  err = hipMalloc((void**)&key_d, sizeof(K)*n);
  CUDA_CHECK(err, "segments: alloc key_d");
  err = hipMalloc((void**)&val_d, sizeof(V)*n);
  CUDA_CHECK(err, "segments: alloc val_d");
  err = hipMalloc((void**)&seg_d, sizeof(int)*length);
  CUDA_CHECK(err, "segments: alloc seg_d");

  err = hipMemcpy(key_d, key, sizeof(K)*n, hipMemcpyHostToDevice);
  CUDA_CHECK(err, "segments: copy to key_d");
  err = hipMemcpy(val_d, val, sizeof(V)*n, hipMemcpyHostToDevice);
  CUDA_CHECK(err, "segments: copy to val_d");
  err = hipMemcpy(seg_d, seg, sizeof(int)*length, hipMemcpyHostToDevice);
  CUDA_CHECK(err, "segments: copy to seg_d");

  bb::segments::kv::bb_segsort(key_d, val_d, n, seg_d, length, context);

  err = hipMemcpy(key, key_d, sizeof(K)*n, hipMemcpyDeviceToHost);
  CUDA_CHECK(err, "segments: copy to key_d");
  err = hipMemcpy(val, val_d, sizeof(V)*n, hipMemcpyDeviceToHost);
  CUDA_CHECK(err, "segments: copy to val_d");

  err = hipFree(key_d);
  CUDA_CHECK(err, "segments: free key_d");
  err = hipFree(val_d);
  CUDA_CHECK(err, "segments: free val_d");
  err = hipFree(seg_d);
  CUDA_CHECK(err, "segments: free seg_d");

  return 0;
}

template <class K>
int sort_segments(K* key, int* seg, int n, int length) {
  bb::k::SortContext<K> context_k(n, length);
  return sort_segments(key, seg, n, length, &context_k);
}

template <class K, class V>
int sort_segments(K* key, V* val, int* seg, int n, int length) {
  bb::kv::SortContext<K, V> context_kv(n, length);
  return sort_segments(key, val, seg, n, length, &context_kv);
}

jint sort(jlong keys_ptr, jint key_type, jint rows, jint cols, jlong context_ptr) {
  switch (key_type) {
    case BOOLEAN: {
      bb::matrix::k::bb_segsort((unsigned char *) keys_ptr, rows, cols, (bb::k::SortContext<unsigned char> *) context_ptr);
      break;
    }
    case BYTE: {
      bb::matrix::k::bb_segsort((char *) keys_ptr, rows, cols, (bb::k::SortContext<char> *) context_ptr);
      break;
    }
    case CHAR: {
      bb::matrix::k::bb_segsort((unsigned short *) keys_ptr, rows, cols, (bb::k::SortContext<unsigned short> *) context_ptr);
      break;
    }
    case SHORT: {
      bb::matrix::k::bb_segsort((short *) keys_ptr, rows, cols, (bb::k::SortContext<short> *) context_ptr);
      break;
    }
    case INT: {
      bb::matrix::k::bb_segsort((int *) keys_ptr, rows, cols, (bb::k::SortContext<int> *) context_ptr);
      break;
    }
    case LONG: {
      bb::matrix::k::bb_segsort((long long int *) keys_ptr, rows, cols, (bb::k::SortContext<long long int> *) context_ptr);
      break;
    }
    case FLOAT: {
      bb::matrix::k::bb_segsort((float *) keys_ptr, rows, cols, (bb::k::SortContext<float> *) context_ptr);
      break;
    }
    case DOUBLE: {
      bb::matrix::k::bb_segsort((double *) keys_ptr, rows, cols, (bb::k::SortContext<double> *) context_ptr);
      break;
    }
    default:
      return JNI_EINVAL;
  }
  return JNI_OK;
}

jint sort(jlong keys_ptr, jint key_type, jint keys_length, jlong segments_ptr, jint segments_length, jlong context_ptr) {
  switch (key_type) {
    case BOOLEAN: {
      bb::segments::k::bb_segsort((unsigned char *) keys_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::k::SortContext<unsigned char> *) context_ptr);
      break;
    }
    case BYTE: {
      bb::segments::k::bb_segsort((char *) keys_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::k::SortContext<char> *) context_ptr);
      break;
    }
    case CHAR: {
      bb::segments::k::bb_segsort((unsigned short *) keys_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::k::SortContext<unsigned short> *) context_ptr);
      break;
    }
    case SHORT: {
      bb::segments::k::bb_segsort((short *) keys_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::k::SortContext<short> *) context_ptr);
      break;
    }
    case INT: {
      bb::segments::k::bb_segsort((int *) keys_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::k::SortContext<int> *) context_ptr);
      break;
    }
    case LONG: {
      bb::segments::k::bb_segsort((long long int *) keys_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::k::SortContext<long long int> *) context_ptr);
      break;
    }
    case FLOAT: {
      bb::segments::k::bb_segsort((float *) keys_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::k::SortContext<float> *) context_ptr);
      break;
    }
    case DOUBLE: {
      bb::segments::k::bb_segsort((double *) keys_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::k::SortContext<double> *) context_ptr);
      break;
    }
    default:
      return JNI_EINVAL;
  }
  return JNI_OK;
}

jint sort(jlong keys_ptr, jint key_type, jlong values_ptr, jint value_type, jint rows, jint cols, jlong context_ptr) {
  switch (key_type) {
    case BOOLEAN: {
      switch (value_type) {
        case BOOLEAN:
          bb::matrix::kv::bb_segsort((unsigned char *) keys_ptr, (unsigned char *) values_ptr, rows, cols, (bb::kv::SortContext<unsigned char, unsigned char> *) context_ptr);
          break;
        case BYTE:
          bb::matrix::kv::bb_segsort((unsigned char *) keys_ptr, (char *) values_ptr, rows, cols, (bb::kv::SortContext<unsigned char, char> *) context_ptr);
          break;
        case CHAR:
          bb::matrix::kv::bb_segsort((unsigned char *) keys_ptr, (unsigned short *) values_ptr, rows, cols, (bb::kv::SortContext<unsigned char, unsigned short> *) context_ptr);
          break;
        case SHORT:
          bb::matrix::kv::bb_segsort((unsigned char *) keys_ptr, (short *) values_ptr, rows, cols, (bb::kv::SortContext<unsigned char, short> *) context_ptr);
          break;
        case INT:
          bb::matrix::kv::bb_segsort((unsigned char *) keys_ptr, (int *) values_ptr, rows, cols, (bb::kv::SortContext<unsigned char, int> *) context_ptr);
          break;
        case LONG:
          bb::matrix::kv::bb_segsort((unsigned char *) keys_ptr, (long long int *) values_ptr, rows, cols, (bb::kv::SortContext<unsigned char, long long int> *) context_ptr);
          break;
        case FLOAT:
          bb::matrix::kv::bb_segsort((unsigned char *) keys_ptr, (float *) values_ptr, rows, cols, (bb::kv::SortContext<unsigned char, float> *) context_ptr);
          break;
        case DOUBLE:
          bb::matrix::kv::bb_segsort((unsigned char *) keys_ptr, (double *) values_ptr, rows, cols, (bb::kv::SortContext<unsigned char, double> *) context_ptr);
          break;
        default:
          return JNI_EINVAL;
      }
      break;
    }
    case BYTE: {
      switch (value_type) {
        case BOOLEAN:
          bb::matrix::kv::bb_segsort((char *) keys_ptr, (unsigned char *) values_ptr, rows, cols, (bb::kv::SortContext<char, unsigned char> *) context_ptr);
          break;
        case BYTE:
          bb::matrix::kv::bb_segsort((char *) keys_ptr, (char *) values_ptr, rows, cols, (bb::kv::SortContext<char, char> *) context_ptr);
          break;
        case CHAR:
          bb::matrix::kv::bb_segsort((char *) keys_ptr, (unsigned short *) values_ptr, rows, cols, (bb::kv::SortContext<char, unsigned short> *) context_ptr);
          break;
        case SHORT:
          bb::matrix::kv::bb_segsort((char *) keys_ptr, (short *) values_ptr, rows, cols, (bb::kv::SortContext<char, short> *) context_ptr);
          break;
        case INT:
          bb::matrix::kv::bb_segsort((char *) keys_ptr, (int *) values_ptr, rows, cols, (bb::kv::SortContext<char, int> *) context_ptr);
          break;
        case LONG:
          bb::matrix::kv::bb_segsort((char *) keys_ptr, (long long int *) values_ptr, rows, cols, (bb::kv::SortContext<char, long long int> *) context_ptr);
          break;
        case FLOAT:
          bb::matrix::kv::bb_segsort((char *) keys_ptr, (float *) values_ptr, rows, cols, (bb::kv::SortContext<char, float> *) context_ptr);
          break;
        case DOUBLE:
          bb::matrix::kv::bb_segsort((char *) keys_ptr, (double *) values_ptr, rows, cols, (bb::kv::SortContext<char, double> *) context_ptr);
          break;
        default:
          return JNI_EINVAL;
      }
      break;
    }
    case CHAR: {
      switch (value_type) {
        case BOOLEAN:
          bb::matrix::kv::bb_segsort((unsigned short *) keys_ptr, (unsigned char *) values_ptr, rows, cols, (bb::kv::SortContext<unsigned short, unsigned char> *) context_ptr);
          break;
        case BYTE:
          bb::matrix::kv::bb_segsort((unsigned short *) keys_ptr, (char *) values_ptr, rows, cols, (bb::kv::SortContext<unsigned short, char> *) context_ptr);
          break;
        case CHAR:
          bb::matrix::kv::bb_segsort((unsigned short *) keys_ptr, (unsigned short *) values_ptr, rows, cols, (bb::kv::SortContext<unsigned short, unsigned short> *) context_ptr);
          break;
        case SHORT:
          bb::matrix::kv::bb_segsort((unsigned short *) keys_ptr, (short *) values_ptr, rows, cols, (bb::kv::SortContext<unsigned short, short> *) context_ptr);
          break;
        case INT:
          bb::matrix::kv::bb_segsort((unsigned short *) keys_ptr, (int *) values_ptr, rows, cols, (bb::kv::SortContext<unsigned short, int> *) context_ptr);
          break;
        case LONG:
          bb::matrix::kv::bb_segsort((unsigned short *) keys_ptr, (long long int *) values_ptr, rows, cols, (bb::kv::SortContext<unsigned short, long long int> *) context_ptr);
          break;
        case FLOAT:
          bb::matrix::kv::bb_segsort((unsigned short *) keys_ptr, (float *) values_ptr, rows, cols, (bb::kv::SortContext<unsigned short, float> *) context_ptr);
          break;
        case DOUBLE:
          bb::matrix::kv::bb_segsort((unsigned short *) keys_ptr, (double *) values_ptr, rows, cols, (bb::kv::SortContext<unsigned short, double> *) context_ptr);
          break;
        default:
          return JNI_EINVAL;
      }
      break;
    }
    case SHORT: {
      switch (value_type) {
        case BOOLEAN:
          bb::matrix::kv::bb_segsort((short *) keys_ptr, (unsigned char *) values_ptr, rows, cols, (bb::kv::SortContext<short, unsigned char> *) context_ptr);
          break;
        case BYTE:
          bb::matrix::kv::bb_segsort((short *) keys_ptr, (char *) values_ptr, rows, cols, (bb::kv::SortContext<short, char> *) context_ptr);
          break;
        case CHAR:
          bb::matrix::kv::bb_segsort((short *) keys_ptr, (unsigned short *) values_ptr, rows, cols, (bb::kv::SortContext<short, unsigned short> *) context_ptr);
          break;
        case SHORT:
          bb::matrix::kv::bb_segsort((short *) keys_ptr, (short *) values_ptr, rows, cols, (bb::kv::SortContext<short, short> *) context_ptr);
          break;
        case INT:
          bb::matrix::kv::bb_segsort((short *) keys_ptr, (int *) values_ptr, rows, cols, (bb::kv::SortContext<short, int> *) context_ptr);
          break;
        case LONG:
          bb::matrix::kv::bb_segsort((short *) keys_ptr, (long long int *) values_ptr, rows, cols, (bb::kv::SortContext<short, long long int> *) context_ptr);
          break;
        case FLOAT:
          bb::matrix::kv::bb_segsort((short *) keys_ptr, (float *) values_ptr, rows, cols, (bb::kv::SortContext<short, float> *) context_ptr);
          break;
        case DOUBLE:
          bb::matrix::kv::bb_segsort((short *) keys_ptr, (double *) values_ptr, rows, cols, (bb::kv::SortContext<short, double> *) context_ptr);
          break;
        default:
          return JNI_EINVAL;
      }
      break;
    }
    case INT: {
      switch (value_type) {
        case BOOLEAN:
          bb::matrix::kv::bb_segsort((int *) keys_ptr, (unsigned char *) values_ptr, rows, cols, (bb::kv::SortContext<int, unsigned char> *) context_ptr);
          break;
        case BYTE:
          bb::matrix::kv::bb_segsort((int *) keys_ptr, (char *) values_ptr, rows, cols, (bb::kv::SortContext<int, char> *) context_ptr);
          break;
        case CHAR:
          bb::matrix::kv::bb_segsort((int *) keys_ptr, (unsigned short *) values_ptr, rows, cols, (bb::kv::SortContext<int, unsigned short> *) context_ptr);
          break;
        case SHORT:
          bb::matrix::kv::bb_segsort((int *) keys_ptr, (short *) values_ptr, rows, cols, (bb::kv::SortContext<int, short> *) context_ptr);
          break;
        case INT:
          bb::matrix::kv::bb_segsort((int *) keys_ptr, (int *) values_ptr, rows, cols, (bb::kv::SortContext<int, int> *) context_ptr);
          break;
        case LONG:
          bb::matrix::kv::bb_segsort((int *) keys_ptr, (long long int *) values_ptr, rows, cols, (bb::kv::SortContext<int, long long int> *) context_ptr);
          break;
        case FLOAT:
          bb::matrix::kv::bb_segsort((int *) keys_ptr, (float *) values_ptr, rows, cols, (bb::kv::SortContext<int, float> *) context_ptr);
          break;
        case DOUBLE:
          bb::matrix::kv::bb_segsort((int *) keys_ptr, (double *) values_ptr, rows, cols, (bb::kv::SortContext<int, double> *) context_ptr);
          break;
        default:
          return JNI_EINVAL;
      }
      break;
    }
    case LONG: {
      switch (value_type) {
        case BOOLEAN:
          bb::matrix::kv::bb_segsort((long long int *) keys_ptr, (unsigned char *) values_ptr, rows, cols, (bb::kv::SortContext<long long int, unsigned char> *) context_ptr);
          break;
        case BYTE:
          bb::matrix::kv::bb_segsort((long long int *) keys_ptr, (char *) values_ptr, rows, cols, (bb::kv::SortContext<long long int, char> *) context_ptr);
          break;
        case CHAR:
          bb::matrix::kv::bb_segsort((long long int *) keys_ptr, (unsigned short *) values_ptr, rows, cols, (bb::kv::SortContext<long long int, unsigned short> *) context_ptr);
          break;
        case SHORT:
          bb::matrix::kv::bb_segsort((long long int *) keys_ptr, (short *) values_ptr, rows, cols, (bb::kv::SortContext<long long int, short> *) context_ptr);
          break;
        case INT:
          bb::matrix::kv::bb_segsort((long long int *) keys_ptr, (int *) values_ptr, rows, cols, (bb::kv::SortContext<long long int, int> *) context_ptr);
          break;
        case LONG:
          bb::matrix::kv::bb_segsort((long long int *) keys_ptr, (long long int *) values_ptr, rows, cols, (bb::kv::SortContext<long long int, long long int> *) context_ptr);
          break;
        case FLOAT:
          bb::matrix::kv::bb_segsort((long long int *) keys_ptr, (float *) values_ptr, rows, cols, (bb::kv::SortContext<long long int, float> *) context_ptr);
          break;
        case DOUBLE:
          bb::matrix::kv::bb_segsort((long long int *) keys_ptr, (double *) values_ptr, rows, cols, (bb::kv::SortContext<long long int, double> *) context_ptr);
          break;
        default:
          return JNI_EINVAL;
      }
      break;
    }
    case FLOAT: {
      switch (value_type) {
        case BOOLEAN:
          bb::matrix::kv::bb_segsort((float *) keys_ptr, (unsigned char *) values_ptr, rows, cols, (bb::kv::SortContext<float, unsigned char> *) context_ptr);
          break;
        case BYTE:
          bb::matrix::kv::bb_segsort((float *) keys_ptr, (char *) values_ptr, rows, cols, (bb::kv::SortContext<float, char> *) context_ptr);
          break;
        case CHAR:
          bb::matrix::kv::bb_segsort((float *) keys_ptr, (unsigned short *) values_ptr, rows, cols, (bb::kv::SortContext<float, unsigned short> *) context_ptr);
          break;
        case SHORT:
          bb::matrix::kv::bb_segsort((float *) keys_ptr, (short *) values_ptr, rows, cols, (bb::kv::SortContext<float, short> *) context_ptr);
          break;
        case INT:
          bb::matrix::kv::bb_segsort((float *) keys_ptr, (int *) values_ptr, rows, cols, (bb::kv::SortContext<float, int> *) context_ptr);
          break;
        case LONG:
          bb::matrix::kv::bb_segsort((float *) keys_ptr, (long long int *) values_ptr, rows, cols, (bb::kv::SortContext<float, long long int> *) context_ptr);
          break;
        case FLOAT:
          bb::matrix::kv::bb_segsort((float *) keys_ptr, (float *) values_ptr, rows, cols, (bb::kv::SortContext<float, float> *) context_ptr);
          break;
        case DOUBLE:
          bb::matrix::kv::bb_segsort((float *) keys_ptr, (double *) values_ptr, rows, cols, (bb::kv::SortContext<float, double> *) context_ptr);
          break;
        default:
          return JNI_EINVAL;
      }
      break;
    }
    case DOUBLE: {
      switch (value_type) {
        case BOOLEAN:
          bb::matrix::kv::bb_segsort((double *) keys_ptr, (unsigned char *) values_ptr, rows, cols, (bb::kv::SortContext<double, unsigned char> *) context_ptr);
          break;
        case BYTE:
          bb::matrix::kv::bb_segsort((double *) keys_ptr, (char *) values_ptr, rows, cols, (bb::kv::SortContext<double, char> *) context_ptr);
          break;
        case CHAR:
          bb::matrix::kv::bb_segsort((double *) keys_ptr, (unsigned short *) values_ptr, rows, cols, (bb::kv::SortContext<double, unsigned short> *) context_ptr);
          break;
        case SHORT:
          bb::matrix::kv::bb_segsort((double *) keys_ptr, (short *) values_ptr, rows, cols, (bb::kv::SortContext<double, short> *) context_ptr);
          break;
        case INT:
          bb::matrix::kv::bb_segsort((double *) keys_ptr, (int *) values_ptr, rows, cols, (bb::kv::SortContext<double, int> *) context_ptr);
          break;
        case LONG:
          bb::matrix::kv::bb_segsort((double *) keys_ptr, (long long int *) values_ptr, rows, cols, (bb::kv::SortContext<double, long long int> *) context_ptr);
          break;
        case FLOAT:
          bb::matrix::kv::bb_segsort((double *) keys_ptr, (float *) values_ptr, rows, cols, (bb::kv::SortContext<double, float> *) context_ptr);
          break;
        case DOUBLE:
          bb::matrix::kv::bb_segsort((double *) keys_ptr, (double *) values_ptr, rows, cols, (bb::kv::SortContext<double, double> *) context_ptr);
          break;
        default:
          return JNI_EINVAL;
      }
      break;
    }
    default:
      return JNI_EINVAL;
  }
  return JNI_OK;
}

jint sort(jlong keys_ptr, jint key_type, jlong values_ptr, jint value_type, jint keys_length, jlong segments_ptr, jint segments_length, jlong context_ptr) {
  switch (key_type) {
    case BOOLEAN: {
      switch (value_type) {
        case BOOLEAN:
          bb::segments::kv::bb_segsort((unsigned char *) keys_ptr, (unsigned char *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<unsigned char, unsigned char> *) context_ptr);
          break;
        case BYTE:
          bb::segments::kv::bb_segsort((unsigned char *) keys_ptr, (char *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<unsigned char, char> *) context_ptr);
          break;
        case CHAR:
          bb::segments::kv::bb_segsort((unsigned char *) keys_ptr, (unsigned short *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<unsigned char, unsigned short> *) context_ptr);
          break;
        case SHORT:
          bb::segments::kv::bb_segsort((unsigned char *) keys_ptr, (short *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<unsigned char, short> *) context_ptr);
          break;
        case INT:
          bb::segments::kv::bb_segsort((unsigned char *) keys_ptr, (int *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<unsigned char, int> *) context_ptr);
          break;
        case LONG:
          bb::segments::kv::bb_segsort((unsigned char *) keys_ptr, (long long int *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<unsigned char, long long int> *) context_ptr);
          break;
        case FLOAT:
          bb::segments::kv::bb_segsort((unsigned char *) keys_ptr, (float *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<unsigned char, float> *) context_ptr);
          break;
        case DOUBLE:
          bb::segments::kv::bb_segsort((unsigned char *) keys_ptr, (double *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<unsigned char, double> *) context_ptr);
          break;
        default:
          return JNI_EINVAL;
      }
      break;
    }
    case BYTE: {
      switch (value_type) {
        case BOOLEAN:
          bb::segments::kv::bb_segsort((char *) keys_ptr, (unsigned char *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<char, unsigned char> *) context_ptr);
          break;
        case BYTE:
          bb::segments::kv::bb_segsort((char *) keys_ptr, (char *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<char, char> *) context_ptr);
          break;
        case CHAR:
          bb::segments::kv::bb_segsort((char *) keys_ptr, (unsigned short *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<char, unsigned short> *) context_ptr);
          break;
        case SHORT:
          bb::segments::kv::bb_segsort((char *) keys_ptr, (short *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<char, short> *) context_ptr);
          break;
        case INT:
          bb::segments::kv::bb_segsort((char *) keys_ptr, (int *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<char, int> *) context_ptr);
          break;
        case LONG:
          bb::segments::kv::bb_segsort((char *) keys_ptr, (long long int *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<char, long long int> *) context_ptr);
          break;
        case FLOAT:
          bb::segments::kv::bb_segsort((char *) keys_ptr, (float *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<char, float> *) context_ptr);
          break;
        case DOUBLE:
          bb::segments::kv::bb_segsort((char *) keys_ptr, (double *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<char, double> *) context_ptr);
          break;
        default:
          return JNI_EINVAL;
      }
      break;
    }
    case CHAR: {
      switch (value_type) {
        case BOOLEAN:
          bb::segments::kv::bb_segsort((unsigned short *) keys_ptr, (unsigned char *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<unsigned short, unsigned char> *) context_ptr);
          break;
        case BYTE:
          bb::segments::kv::bb_segsort((unsigned short *) keys_ptr, (char *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<unsigned short, char> *) context_ptr);
          break;
        case CHAR:
          bb::segments::kv::bb_segsort((unsigned short *) keys_ptr, (unsigned short *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<unsigned short, unsigned short> *) context_ptr);
          break;
        case SHORT:
          bb::segments::kv::bb_segsort((unsigned short *) keys_ptr, (short *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<unsigned short, short> *) context_ptr);
          break;
        case INT:
          bb::segments::kv::bb_segsort((unsigned short *) keys_ptr, (int *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<unsigned short, int> *) context_ptr);
          break;
        case LONG:
          bb::segments::kv::bb_segsort((unsigned short *) keys_ptr, (long long int *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<unsigned short, long long int> *) context_ptr);
          break;
        case FLOAT:
          bb::segments::kv::bb_segsort((unsigned short *) keys_ptr, (float *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<unsigned short, float> *) context_ptr);
          break;
        case DOUBLE:
          bb::segments::kv::bb_segsort((unsigned short *) keys_ptr, (double *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<unsigned short, double> *) context_ptr);
          break;
        default:
          return JNI_EINVAL;
      }
      break;
    }
    case SHORT: {
      switch (value_type) {
        case BOOLEAN:
          bb::segments::kv::bb_segsort((short *) keys_ptr, (unsigned char *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<short, unsigned char> *) context_ptr);
          break;
        case BYTE:
          bb::segments::kv::bb_segsort((short *) keys_ptr, (char *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<short, char> *) context_ptr);
          break;
        case CHAR:
          bb::segments::kv::bb_segsort((short *) keys_ptr, (unsigned short *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<short, unsigned short> *) context_ptr);
          break;
        case SHORT:
          bb::segments::kv::bb_segsort((short *) keys_ptr, (short *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<short, short> *) context_ptr);
          break;
        case INT:
          bb::segments::kv::bb_segsort((short *) keys_ptr, (int *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<short, int> *) context_ptr);
          break;
        case LONG:
          bb::segments::kv::bb_segsort((short *) keys_ptr, (long long int *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<short, long long int> *) context_ptr);
          break;
        case FLOAT:
          bb::segments::kv::bb_segsort((short *) keys_ptr, (float *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<short, float> *) context_ptr);
          break;
        case DOUBLE:
          bb::segments::kv::bb_segsort((short *) keys_ptr, (double *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<short, double> *) context_ptr);
          break;
        default:
          return JNI_EINVAL;
      }
      break;
    }
    case INT: {
      switch (value_type) {
        case BOOLEAN:
          bb::segments::kv::bb_segsort((int *) keys_ptr, (unsigned char *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<int, unsigned char> *) context_ptr);
          break;
        case BYTE:
          bb::segments::kv::bb_segsort((int *) keys_ptr, (char *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<int, char> *) context_ptr);
          break;
        case CHAR:
          bb::segments::kv::bb_segsort((int *) keys_ptr, (unsigned short *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<int, unsigned short> *) context_ptr);
          break;
        case SHORT:
          bb::segments::kv::bb_segsort((int *) keys_ptr, (short *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<int, short> *) context_ptr);
          break;
        case INT:
          bb::segments::kv::bb_segsort((int *) keys_ptr, (int *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<int, int> *) context_ptr);
          break;
        case LONG:
          bb::segments::kv::bb_segsort((int *) keys_ptr, (long long int *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<int, long long int> *) context_ptr);
          break;
        case FLOAT:
          bb::segments::kv::bb_segsort((int *) keys_ptr, (float *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<int, float> *) context_ptr);
          break;
        case DOUBLE:
          bb::segments::kv::bb_segsort((int *) keys_ptr, (double *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<int, double> *) context_ptr);
          break;
        default:
          return JNI_EINVAL;
      }
      break;
    }
    case LONG: {
      switch (value_type) {
        case BOOLEAN:
          bb::segments::kv::bb_segsort((long long int *) keys_ptr, (unsigned char *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<long long int, unsigned char> *) context_ptr);
          break;
        case BYTE:
          bb::segments::kv::bb_segsort((long long int *) keys_ptr, (char *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<long long int, char> *) context_ptr);
          break;
        case CHAR:
          bb::segments::kv::bb_segsort((long long int *) keys_ptr, (unsigned short *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<long long int, unsigned short> *) context_ptr);
          break;
        case SHORT:
          bb::segments::kv::bb_segsort((long long int *) keys_ptr, (short *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<long long int, short> *) context_ptr);
          break;
        case INT:
          bb::segments::kv::bb_segsort((long long int *) keys_ptr, (int *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<long long int, int> *) context_ptr);
          break;
        case LONG:
          bb::segments::kv::bb_segsort((long long int *) keys_ptr, (long long int *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<long long int, long long int> *) context_ptr);
          break;
        case FLOAT:
          bb::segments::kv::bb_segsort((long long int *) keys_ptr, (float *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<long long int, float> *) context_ptr);
          break;
        case DOUBLE:
          bb::segments::kv::bb_segsort((long long int *) keys_ptr, (double *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<long long int, double> *) context_ptr);
          break;
        default:
          return JNI_EINVAL;
      }
      break;
    }
    case FLOAT: {
      switch (value_type) {
        case BOOLEAN:
          bb::segments::kv::bb_segsort((float *) keys_ptr, (unsigned char *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<float, unsigned char> *) context_ptr);
          break;
        case BYTE:
          bb::segments::kv::bb_segsort((float *) keys_ptr, (char *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<float, char> *) context_ptr);
          break;
        case CHAR:
          bb::segments::kv::bb_segsort((float *) keys_ptr, (unsigned short *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<float, unsigned short> *) context_ptr);
          break;
        case SHORT:
          bb::segments::kv::bb_segsort((float *) keys_ptr, (short *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<float, short> *) context_ptr);
          break;
        case INT:
          bb::segments::kv::bb_segsort((float *) keys_ptr, (int *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<float, int> *) context_ptr);
          break;
        case LONG:
          bb::segments::kv::bb_segsort((float *) keys_ptr, (long long int *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<float, long long int> *) context_ptr);
          break;
        case FLOAT:
          bb::segments::kv::bb_segsort((float *) keys_ptr, (float *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<float, float> *) context_ptr);
          break;
        case DOUBLE:
          bb::segments::kv::bb_segsort((float *) keys_ptr, (double *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<float, double> *) context_ptr);
          break;
        default:
          return JNI_EINVAL;
      }
      break;
    }
    case DOUBLE: {
      switch (value_type) {
        case BOOLEAN:
          bb::segments::kv::bb_segsort((double *) keys_ptr, (unsigned char *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<double, unsigned char> *) context_ptr);
          break;
        case BYTE:
          bb::segments::kv::bb_segsort((double *) keys_ptr, (char *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<double, char> *) context_ptr);
          break;
        case CHAR:
          bb::segments::kv::bb_segsort((double *) keys_ptr, (unsigned short *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<double, unsigned short> *) context_ptr);
          break;
        case SHORT:
          bb::segments::kv::bb_segsort((double *) keys_ptr, (short *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<double, short> *) context_ptr);
          break;
        case INT:
          bb::segments::kv::bb_segsort((double *) keys_ptr, (int *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<double, int> *) context_ptr);
          break;
        case LONG:
          bb::segments::kv::bb_segsort((double *) keys_ptr, (long long int *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<double, long long int> *) context_ptr);
          break;
        case FLOAT:
          bb::segments::kv::bb_segsort((double *) keys_ptr, (float *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<double, float> *) context_ptr);
          break;
        case DOUBLE:
          bb::segments::kv::bb_segsort((double *) keys_ptr, (double *) values_ptr, keys_length, (int *) segments_ptr, segments_length, (bb::kv::SortContext<double, double> *) context_ptr);
          break;
        default:
          return JNI_EINVAL;
      }
      break;
    }
    default:
      return JNI_EINVAL;
  }
  return JNI_OK;
}